#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>

#define IND(x, y, d) = ((int)((y * d) + x))

using namespace std;
using namespace Eigen;


template <typename T>
T* initVectorField(unsigned dim) {
    T *ret;
    hipMalloc(&ret, dim*dim*sizeof(T));
    hipMemset(ret, T::Zero(), dim*dim);
    return ret;
}


template <typename T>
T* initScalarField(unsigned dim) {
    T *ret;
    hipMalloc(&ret, dim*dim*sizeof(T));
    hipMemset(ret, (T)0, dim*dim);
    return ret;
}

/***
 * Bilinear Interpolation
 * https://en.wikipedia.org/wiki/Bilinear_interpolation
 */
__device__ Vector2f bilerp(Vector2f pos, Vector2f *field, unsigned dim) {
    int i = (int)pos(0);
    int j = (int)pos(1);
    double dx = (double)(pos(0) - i);
    double dy = (double)(pos(1) - j);

    if (i < 0 || i >= dim || j < 0 || j >= dim) {
        // Out of bounds.
        return Vector2f::Zero();
    } else {
        // Perform bilinear interpolation.
        Vector2f f00 = field[IND(x-1, y-1, dim)];
        Vector2f f01 = field[IND(x+1, y-1, dim)];
        Vector2f f10 = field[IND(x-1, y+1, dim)];
        Vector2f f11 = field[IND(x+1, y+1, dim)];
        Vector2f f0 = (1 - dx) * f00 + dx * f10;
        Vector2f f1 = (1 - dx) * f01 + dx * f11;
        return (1 - dy) * f0 + dy * f1;
    }
}


/***
 * x is the coordinate/position vector following notation of chp 38.
 * velfield is u, the velocity field as of the current time quanta.
 * field is the current field being updated.
*/
__device__ void advect(Vector2f x, Vector2f *field, Vector2f *velfield, float timestep, float rdx, unsigned dim) {
    Vector2f pos = x - timestep*rdx*velfield[IND(x(0), x(1), dim)];
    field[IND(x(0), x(1), dim)] = bilerp(pos, field, dim);
}


__device__ void jacobi(Vector2f x, Vector2f *field, float alpha, float beta, Vector2f b) {
    Vector2f f00 = field[IND(x - 1, y - 1, dim)];
    Vector2f f01 = field[IND(x + 1, y - 1, dim)];
    Vector2f f10 = field[IND(x - 1, y + 1, dim)];
    Vector2f f11 = field[IND(x + 1, y + 1, dim)];

    return (f00 + f01 + f10 + f11 + alpha*b) / beta;
}

__global__ void kernel(void) {
    return;
}

int main(void) {
    // dimension of vector fields
    unsigned dim = 1024;
    // resolution of display
    unsigned res = 1024;
    // how many pixels a cell of the vector field represents
    float rdx = res / dim;
    
    Vector2f *dev_velocity = initVectorField<Vector2f>(dim);
    float *dev_pressure = initScalarField<float>(dim);

    kernel<<<1, dim>>>();
    return 0;
}