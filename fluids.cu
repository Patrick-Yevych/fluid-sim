#include "hip/hip_runtime.h"
// A DeerHacks last-hour khuya submission.
// nvcc fluids.cu -o ./out -lglfw -lGLU -lGL
#include "const.h"

// mouse click location
Vector2f C;
// direction and length of mouse drag
Vector2f F;
// decay rate
float global_decay_rate = DECAY_RATE;

/**
 * Initializes a vector or scalar field with initial conditions to both
 * the hostside and deviceside.
 * @param f The field on the host.
 * @param dev_f The field on the device.
 * @param val Initial conditions.
 * @param dim The dimensions [for boundary checking]
 * @authors Patrick Yevych
 */
template <typename T>
void initializeField(T **f, T **dev_f, T val, unsigned dim)
{
    *f = (T *)malloc(dim * dim * sizeof(T));
    hipMalloc(dev_f, dim * dim * sizeof(T));
    for (int i = 0; i < dim * dim; i++)
        *(*f + i) = val;
    hipMemcpy(*dev_f, *f, dim * dim * sizeof(T), hipMemcpyHostToDevice);
}

/**
 * Called whenever a click or release even happens on the window.
 * Updates the convection F and the mouse click location C.
 * @param window The GLFWwindow object to be applied to
 * @param button ID of the clickable clicked
 * @param action The type of action registered
 * @param mods Any specific mods applied to this action
 * @authors Patrick Yevych
 */
void mouse_button_callback(GLFWwindow *window, int button, int action, int mods)
{
    double xpos, ypos, xend, yend;
    if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS) {
        glfwGetCursorPos(window, &xpos, &ypos);
        C = Vector2f((int)xpos, (int)ypos);
    }
    else if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_RELEASE) {
        glfwGetCursorPos(window, &xend, &yend);
        F = Vector2f(xend - C(0), yend - C(1));
    }
}

/**
 * Decays the convection force F.
 * @authors Patrick Yevych
 */
void decayForce()
{
    float nx = F(0) - global_decay_rate;
    float ny = F(1) - global_decay_rate;
    nx = (nx > 0) ? nx : 0;
    ny = (ny > 0) ? ny : 0;
    F = Vector2f(nx, ny);
}

/**
 * Implementation of bilinear interpolation given input location.
 * @param pos The input location, supporting intermediate positions
 * @param field The vector field
 * @param dim The dimensions [for boundary checking]
 * @authors Alex Apostolou, Samaria Mulligan
 * @link https://en.wikipedia.org/wiki/Bilinear_interpolation
 */
__device__ Vector2f bilerp(Vector2f pos, Vector2f *field, unsigned dim)
{
    int i = pos(0);
    int j = pos(1);
    double dx = pos(0) - i;
    double dy = pos(1) - j;

    if (i < 0 || i >= dim || j < 0 || j >= dim) {
        // Out of bounds.
        return Vector2f::Zero();
    }
    else {
        // Perform bilinear interpolation.

        Vector2f f00 = (i - 1 < 0 || i - 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? Vector2f::Zero() : field[IND(i - 1, j - 1, dim)];
        Vector2f f01 = (i + 1 < 0 || i + 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? Vector2f::Zero() : field[IND(i + 1, j - 1, dim)];
        Vector2f f10 = (i - 1 < 0 || i - 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? Vector2f::Zero() : field[IND(i - 1, j + 1, dim)];
        Vector2f f11 = (i + 1 < 0 || i + 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? Vector2f::Zero() : field[IND(i + 1, j + 1, dim)];

        Vector2f f0 = (1 - dx) * f00 + dx * f10;
        Vector2f f1 = (1 - dx) * f01 + dx * f11;

        return (1 - dy) * f0 + dy * f1;
    }
}

/**
 * Obtain the approximate divergence of a vector field.
 * The divergence is calculated using the immediate neighboring value only
 * across the four cardinal directions.
 * @param x Cartesian location of the field
 * @param from The vector field
 * @param halfrdx Half of the rdx value [for efficiency reasons]
 * @param dim The maximum dimension of the field [for bound checking]
 * @return The approximate divergence value
 * @authors Alex Apostolou
 */
__device__ float divergence(Vector2f x, Vector2f *from, float halfrdx, unsigned dim)
{
    int i = x(0);
    int j = x(1);
    if (i < 0 || i >= dim || j < 0 || j >= dim)
        return 0;

    Vector2f wL = (i - 1 < 0) ? Vector2f::Zero() : from[IND(i - 1, j, dim)];
    Vector2f wR = (i + 1 >= dim) ? Vector2f::Zero() : from[IND(i + 1, j, dim)];
    Vector2f wB = (j - 1 < 0) ? Vector2f::Zero() : from[IND(i, j - 1, dim)];
    Vector2f wT = (j + 1 >= dim) ? Vector2f::Zero() : from[IND(i, j + 1, dim)];

    return halfrdx * (wR(0) - wL(0), wT(1) - wB(1));
}

/**
 * Obtain the approximate gradient of a scalar field [in this case, p].
 * The gradient is calculated using the immediate neighboring value only.
 * @param x Cartesian location of the field
 * @param p The scalar field [pressure]
 * @param halfrdx Half of the rdx value [for efficiency reasons]
 * @param dim The maximum dimension of the field [for bound checking]
 * @return The approximate gradient, as a Vector2f
 * @authors Alex Apostolou
 */
__device__ Vector2f gradient(Vector2f x, float *p, float halfrdx, unsigned dim)
{
    int i = x(0);
    int j = x(1);

    if (i < 0 || i >= dim || j < 0 || j >= dim)
        return Vector2f::Zero();

    float pL = (i - 1 < 0) ? 0 : p[IND(i - 1, j, dim)];
    float pR = (i + 1 >= dim) ? 0 : p[IND(i + 1, j, dim)];
    float pB = (j - 1 < 0) ? 0 : p[IND(i, j - 1, dim)];
    float pT = (j + 1 >= dim) ? 0 : p[IND(i, j + 1, dim)];

    return halfrdx * Vector2f(pR - pL, pT - pB);
}

/***
 * Computes the advection of the fluid.
 * @param x is the coordinate/position vector following notation of chp 38.
 * @param velfield is u, the velocity field as of the current time quanta.
 * @param field is the current field being updated.
 * @param timestep delta t for next iteration
 * @param rdx approximation constant
 * @param dim The maximum dimension of the field [for bound checking]
 * @authors Patrick Yevych
 */
__device__ void advect(Vector2f x, Vector2f *field, Vector2f *velfield, float timestep, float rdx, unsigned dim)
{
    Vector2f pos = x - timestep * rdx * velfield[IND(x(0), x(1), dim)];
    field[IND(x(0), x(1), dim)] = bilerp(pos, field, dim);
}

/**
 * Generalized Jacobi for computing pressure or viscous diffusion of fluid.
 * @param x is the coordinate/position vector following notation of paper.
 * @param field The relevant vector field
 * @param alpha rdx*rdx/(viscosity*timestep) for diffusion; -1*timestep*timestep for pressure.
 * @param beta 4+alpha for diffusion; 4 for pressure.
 * @param b u(x) for diffusion; divergence for pressure.
 * @param dim The maximum dimension of the field [for bound checking].
 * @authors Patrick Yevych
 */
template <typename T>
__device__ void jacobi(Vector2f x, T *field, float alpha, float beta, T b, T zero, unsigned dim)
{
    int i = (int)x(0);
    int j = (int)x(1);

    T f00 = (i - 1 < 0 || i - 1 >= dim || j < 0 || j >= dim) ? zero : field[IND(i - 1, j, dim)];
    T f01 = (i + 1 < 0 || i + 1 >= dim || j < 0 || j >= dim) ? zero : field[IND(i + 1, j, dim)];
    T f10 = (i < 0 || i >= dim || j - 1 < 0 || j - 1 >= dim) ? zero : field[IND(i, j - 1, dim)];
    T f11 = (i < 0 || i >= dim || j + 1 < 0 || j + 1 >= dim) ? zero : field[IND(i, j + 1, dim)];
    T ab = (i < 0 || i >= dim || j < 0 || j >= dim) ? zero : alpha * b;

    field[IND(i-1, j, dim)] = (f00 + f01 + f10 + f11 + ab) / beta;
}

/**
 * Apply the external source to the deviceside data
 * @param x is the coordinate/position vector following notation of chp 38.
 * @param field The relevant vector field
 * @param C The center of the applied force
 * @param F The value of the applied force
 * @param timestep The time step per iteration of the program
 * @param r The radius of the applied force
 * @param dim The maximum dimension of the field [for bound checking]
 * @authors Patrick Yevych, Hong Wei, Samaria Mulligan
 */
__device__ void force(Vector2f x, Vector2f *field, Vector2f C, Vector2f F, float timestep, float r, unsigned dim)
{
    float xC[2] = {x(0) - C(0), x(1) - C(1)};
    float exp = (xC[0] * xC[0] + xC[1] * xC[1]) / r;
    int i = x(0);
    int j = x(1);
    Vector2f temp = F * timestep * pow(2.718, exp) * 0.001;
    field[IND(i, j, dim)] += F * timestep * pow(2.718, exp) * 0.001;
    if ((temp(0) != 0 || temp(1) != 0) && x(0) == DIM / 2 && x(1) == DIM / 2)
        printf("G1 = (%f, %f)\n", temp(0), temp(1));
}

/**
 * Navier-Stokes computation kernel.
 * @param u The vector velocity field
 * @param p The scalar pressure field
 * @param rdx Reciprocal of the grid scale
 * @param viscosity The viscosity of the fluid
 * @param C The center of the applied force
 * @param F The value of the applied force
 * @param timestep The time step per iteration of the program
 * @param r The radius of the applied force
 * @param dim The maximum dimension of the field [for bound checking]
 * @authors Patrick Yevych
 */
__global__ void nskernel(Vector2f *u, float *p, float rdx, float viscosity, Vector2f C, Vector2f F, float timestep, float r, unsigned dim)
{
    Vector2f x(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y);

    // advection
    advect(x, u, u, timestep, rdx, dim);
    __syncthreads();

    // diffusion
    float alpha = rdx * rdx / (viscosity * timestep), beta = 4 + alpha;
    jacobi<Vector2f>(x, u, alpha, beta, u[IND(x(0), x(1), dim)], Vector2f::Zero(), dim);
    __syncthreads();

    // force application
    force(x, u, C, F, timestep, r, dim);
    __syncthreads();

    // pressure
    alpha = -1 * rdx * rdx; beta = 4;
    jacobi<float>(x, p, alpha, beta, divergence(x, u, (float)(rdx / 2), dim), 0, dim);
    __syncthreads();

    // u = w - nabla p
    u[IND(x(0), x(1), dim)] -= gradient(x, p, (float)(rdx / 2), dim);
    __syncthreads();

    // print state
    if (x(0) == DIM / 2 && x(1) == DIM / 2)
        printf("u[%.1f, %.1f] = (%f, %f)\n", x(0), x(1), u[IND(x(0), x(1), dim)](0), u[IND(x(0), x(1), dim)](1));
}

/**
 * Given the value of x, obtain corresponding RGB value, for visualization.
 * Adapted from Yuki Koyama.
 * @param x The corresponding intermediate value
 * @authors Hong Wei, Alex Apostolou
 * @link https://github.com/yuki-koyama/tinycolormap
 */
__device__ Vector3f getColor(double x)
{
    double data[][3] = VIRIDIS;

    const double a = CLAMP(x) * 255;
    const double i = std::floor(a);
    const double t = a - i;
    auto d0 = data[static_cast<std::size_t>(std::ceil(a))];
    Vector3f c0(d0[0], d0[1], d0[2]);
    auto d1 = data[static_cast<std::size_t>(std::ceil(a))];
    Vector3f c1(d1[0], d1[1], d1[2]);

    return (1.0 - t) * c0 + t * c1;
}

/**
 * Maps velocity vectors to a color
 * @param uc Array of RGB values for every pixel
 * @param u The velocity vector at that location
 * @param dim The maximum dimension of the field [for bound checking]
 * @authors Patrick Yevych
 */
__global__ void clrkernel(Vector3f *uc, Vector2f *u, unsigned dim)
{
    Vector2f x(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y);
    uc[IND(x(0), x(1), dim)] = getColor(
        (double)u[IND(x(0), x(1), dim)].norm());
}

/**
 * Driver code containing the CUDA kernels and OpenGL rendering.
 * @authors Patrick Yevych, Hong Wei
 */
int main(int argc, char **argv)
{
    // quarter of second timestep
    float timestep = TIMESTEP;
    // dimension of vector fields
    unsigned dim = DIM;
    // resolution of display
    unsigned res = RES;
    // how many pixels a cell of the vector field represents
    float rdx = res / dim;
    // fluid parameters
    float viscosity = VISCOSITY;
    // force decay rate
    global_decay_rate = DECAY_RATE;
    // force radius
    float r = RADIUS;

    // user provided simulation parameters
    if (argc == 5) {
        timestep = atof(argv[1]);
        viscosity = atof(argv[2]);
        global_decay_rate = atof(argv[3]);
        r = atof(argv[4]);
    }
    else if (argc != 1) {
        printf("USAGE: ./out TIMESTEP VISCOSITY DECAY RADIUS\n");
        return 1;
    }

    // force parameters
    C = Vector2f::Zero(); F = Vector2f::Zero();

    // fluid state representation:
    // velocity vector field (u) and pressure scalar field (p).
    Vector2f *u, *dev_u;
    float *p, *dev_p;

    initializeField<Vector2f>(&u, &dev_u, Vector2f::Zero(), dim);
    initializeField<float>(&p, &dev_p, 0, dim);

    // color maps
    Vector3f *uc, *dev_uc;
    initializeField<Vector3f>(&uc, &dev_uc, Vector3f::Zero(), dim);

    // Initialize GLFW
    if (!glfwInit())
        return -1;

    // Create a window
    GLFWwindow *window = glfwCreateWindow(dim, dim, "sim", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }
    // Make the window's context current
    glfwMakeContextCurrent(window);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    // Setup the projection matrix
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, dim, 0, dim, -1, 1);

    // Set up the modelview matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // Load the texture from data
    GLuint tex;
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, dim, dim, 0, GL_RGB, GL_FLOAT, uc);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

    // Enable texturing
    glEnable(GL_TEXTURE_2D);

    // Set the texture as the current texture
    glBindTexture(GL_TEXTURE_2D, tex);

    // Set the texture environment parameters
    glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
    dim3 blocks(dim / BLOCKSIZEX, dim / BLOCKSIZEY);
    // Loop until the user closes
    while (!glfwWindowShouldClose(window)) {

        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, dim, dim, 0, GL_RGB, GL_FLOAT, uc);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, tex);

        glClear(GL_COLOR_BUFFER_BIT);
        // Draw a quad with texture coordinates
        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 0.0f);
        glVertex2i(0, 0);
        glTexCoord2f(1.0f, 0.0f);
        glVertex2i(dim, 0);
        glTexCoord2f(1.0f, 1.0f);
        glVertex2i(dim, dim);
        glTexCoord2f(0.0f, 1.0f);
        glVertex2i(0, dim);
        glEnd();

        // Swap front and back buffers
        glfwSwapBuffers(window);

        // Poll for and process events
        glfwPollEvents();

        // compute navier-stokes and colorize
        nskernel<<<blocks, threads>>>(dev_u, dev_p, rdx, viscosity, C, F, timestep, r, dim);
        hipDeviceSynchronize();
        clrkernel<<<blocks, threads>>>(dev_uc, dev_u, dim);
        hipDeviceSynchronize();
        hipMemcpy(uc, dev_uc, dim * dim * sizeof(Vector3f), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        decayForce();
    }

    free(u);
    free(p);
    free(uc);
    hipFree(dev_u);
    hipFree(dev_p);
    hipFree(dev_uc);

    glfwTerminate();
}