#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>

#define IND(x, y, d) = ((int)((y * d) + x))

using namespace std;
using namespace Eigen;


template <typename T>
T* initVectorField(unsigned dim) {
    T *ret;
    hipMalloc(&ret, dim*dim*sizeof(T));
    hipMemset(ret, T::Zero(), dim*dim);
    return ret;
}


template <typename T>
T* initScalarField(unsigned dim) {
    T *ret;
    hipMalloc(&ret, dim*dim*sizeof(T));
    hipMemset(ret, (T)0, dim*dim);
    return ret;
}

/***
 * Bilinear Interpolation
 * https://en.wikipedia.org/wiki/Bilinear_interpolation
 */
__device__ Vector2f bilerp(Vector2f pos, Vector2f *field, unsigned dim) {
    int i = (int)pos(0);
    int j = (int)pos(1);
    double dx = (double)(pos(0) - i);
    double dy = (double)(pos(1) - j);

    if (i < 0 || i >= dim || j < 0 || j >= dim) {
        // Out of bounds.
        return Vector2f::Zero();
    } else {
        // Perform bilinear interpolation.
        Vector2f f00 = (i - 1 < 0 || i - 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? 0 : field[IND(i - 1, j - 1, dim)];

        Vector2f f01 = (i + 1 < 0 || i + 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? 0 : field[IND(i + 1, j - 1, dim)];

        Vector2f f10 = (i - 1 < 0 || i - 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? 0 : field[IND(i - 1, j + 1, dim)];

        Vector2f f11 = (i + 1 < 0 || i + 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? 0 : field[IND(i + 1, j + 1, dim)];

        Vector2f f0 = (1 - dx) * f00 + dx * f10;
        Vector2f f1 = (1 - dx) * f01 + dx * f11;
        return (1 - dy) * f0 + dy * f1;
    }
}

__device__ void divergence(
    Vector2f x, Vector2f* feild, float halfrdx, unsigned dim)
{
    i = (int)x(0);
    j = (int)x(1);
    if (i < 0 || i >= dim || j < 0 || j >= dim)
        return Vector2f::Zero();

    Vector2f wL = (i - 1 < 0)    ? Vector2f::Zero() : field[IND(i - 1, j, dim)];
    Vector2f wR = (i + 1 >= dim) ? Vector2f::Zero() : field[IND(i + 1, j, dim)];
    Vector2f wB = (j - 1 < 0)    ? Vector2f::Zero() : field[IND(i, j - 1, dim)];
    Vector2f wT = (j + 1 <= dim) ? Vector2f::Zero() : field[IND(i, j + 1, dim)];

    div = halfrdx * ((wR(0) - wL(0))) + (wT(1) - wB(1));

    return div;
}


/***
 * Computes the advection of the fluid.
 * 
 * x is the coordinate/position vector following notation of chp 38.
 * velfield is u, the velocity field as of the current time quanta.
 * field is the current field being updated.
*/
__device__ void advect(Vector2f x, Vector2f *field, Vector2f *velfield, float timestep, float rdx, unsigned dim) {
    Vector2f pos = x - timestep*rdx*velfield[IND(x(0), x(1), dim)];
    field[IND(x(0), x(1), dim)] = bilerp(pos, field, dim);
}

/***
 * Jacobi iteration for computing pressure and
 * viscous diffusion of fluid.
*/
template <typename T>
__device__ void jacobi(Vector2f x, T *field, float alpha, float beta, T *b, unsigned dim) {
    int i = (int)x(0);
    int j = (int)x(1);

    T f00 = (i - 1 < 0 || i - 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? 0 : field[IND(i - 1, j - 1, dim)];

    T f01 = (i + 1 < 0 || i + 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? 0 : field[IND(i + 1, j - 1, dim)];

    T f10 = (i - 1 < 0 || i - 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? 0 : field[IND(i - 1, j + 1, dim)];

    T f11 = (i + 1 < 0 || i + 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? 0 : field[IND(i + 1, j + 1, dim)];

    field[IND(i, j, dim)] = (f00 + f01 + f10 + f11 + alpha*b[IND(i, j, dim)]) / beta;
}

__global__ void kernel(Vector2f *u, Vector2f *p, float timestep, float rdx, float dim, float viscosity) {
    Vector2f x(threadIdx.x, threadIdx.y);
    advect(x, u, u, timestep, rdx, dim);
    //diffusion
    float alpha = (rdx*rdx)/(viscosity*timestep), beta = 4 + alpha;
    jacobi<Vector2f>(x, u, alpha, beta, u, dim);

    return;
}

int main(void) {
    // dimension of vector fields
    unsigned dim = 1024;
    // resolution of display
    unsigned res = 1024;
    // how many pixels a cell of the vector field represents
    float rdx = res / dim;
    
    Vector2f *dev_velocity = initVectorField<Vector2f>(dim); //u


    float *dev_pressure = initScalarField<float>(dim);

    // Iterate
    /*
    u = advect(u);
    u = next_diffusion(u, dx, nu, dt, dim);
    u = addForces(u);

    // Now apply the projection operator to the result.
    p = next_poisson(p, div_w, dx, dim);
    u = subtractPressureGradient(u, p);

    
    
    */

    dim3 block(dim, dim);
    kernel<<<1, block>>>();
    return 0;
}