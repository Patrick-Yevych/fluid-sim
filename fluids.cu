#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#if defined(_WIN32)
#include <windows.h>
#else
#include <unistd.h> // for sleep function. use window.h for windows.
#endif
#include <eigen3/Eigen/Dense>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define IND(x, y, d) int((y) * (d) + (x))

using namespace std;
using Eigen::Vector2f;
using Eigen::Vector2d;

/***
 * Bilinear Interpolation
 * https://en.wikipedia.org/wiki/Bilinear_interpolation
 */
__device__ Vector2f bilerp(Vector2f pos, Vector2f* field, unsigned dim) {
    int i = pos(0);
    int j = pos(1);
    double dx = pos(0) - i;
    double dy = pos(1) - j;

    if (i < 0 || i >= dim || j < 0 || j >= dim) {
        // Out of bounds.
        return Vector2f::Zero();
    }
    else {
        // Perform bilinear interpolation.

        Vector2f f00 = (i - 1 < 0 || i - 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? Vector2f::Zero() : field[IND(i - 1, j - 1, dim)];

        Vector2f f01 = (i + 1 < 0 || i + 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? Vector2f::Zero() : field[IND(i + 1, j - 1, dim)];

        Vector2f f10 = (i - 1 < 0 || i - 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? Vector2f::Zero() : field[IND(i - 1, j + 1, dim)];

        Vector2f f11 = (i + 1 < 0 || i + 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? Vector2f::Zero() : field[IND(i + 1, j + 1, dim)];

        Vector2f f0 = (1 - dx) * f00 + dx * f10;
        Vector2f f1 = (1 - dx) * f01 + dx * f11;
        return (1 - dy) * f0 + dy * f1;
    }
}

__device__ float divergence(
    Vector2f x, Vector2f* from, float halfrdx, unsigned dim)
{
    int i = x(0);
    int j = x(1);
    if (i < 0 || i >= dim || j < 0 || j >= dim)
        return Vector2f::Zero();

    Vector2f wL = (i - 1 < 0) ? Vector2f::Zero() : from[IND(i - 1, j, dim)];
    Vector2f wR = (i + 1 >= dim) ? Vector2f::Zero() : from[IND(i + 1, j, dim)];
    Vector2f wB = (j - 1 < 0) ? Vector2f::Zero() : from[IND(i, j - 1, dim)];
    Vector2f wT = (j + 1 <= dim) ? Vector2f::Zero() : from[IND(i, j + 1, dim)];

    return halfrdx * (wR(0) - wL(0), wT(1) - wB(1);
}


/***
 * only for computing gradient of p.
*/
__device__ Vector2f gradient(
    Vector2f x, float* p, float halfrdx, unsigned dim) {
    int i = x(0);
    int j = x(1);

    if (i < 0 || i >= dim || j < 0 || j >= dim)
        return Vector2f::Zero();

    float pL = (i - 1 < 0)    ? 0 : p[IND(i - 1, j, dim)];
    float pR = (i + 1 >= dim) ? 0 : p[IND(i + 1, j, dim)];
    float pB = (j - 1 < 0)    ? 0 : p[IND(i, j - 1, dim)];
    float pT = (j + 1 >= dim) ? 0 : p[IND(i, j + 1, dim)];

    return halfrdx * Vector2f(pR - pL, pT - pB);
}


/***
 * Computes the advection of the fluid.
 *
 * x is the coordinate/position vector following notation of chp 38.
 * velfield is u, the velocity field as of the current time quanta.
 * field is the current field being updated.
*/
__device__ void advect(Vector2f x, Vector2f* field, Vector2f* velfield, float timestep, float rdx, unsigned dim) {
    Vector2f pos = x - timestep * rdx * velfield[IND(x(0), x(1), dim)];
    field[IND(x(0), x(1), dim)] = bilerp(pos, field, dim);
}

/***
 * Jacobi iteration for computing pressure and
 * viscous diffusion of fluid.
*/
template <typename T>
__device__ void jacobi(Vector2f x, T* field, float alpha, float beta, T b, T zero, unsigned dim) {
    int i = (int)x(0);
    int j = (int)x(1);

    T f00 = (i - 1 < 0 || i - 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? zero : field[IND(i - 1, j - 1, dim)];

    T f01 = (i + 1 < 0 || i + 1 >= dim || j - 1 < 0 || j - 1 >= dim) ? zero : field[IND(i + 1, j - 1, dim)];

    T f10 = (i - 1 < 0 || i - 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? zero : field[IND(i - 1, j + 1, dim)];

    T f11 = (i + 1 < 0 || i + 1 >= dim || j + 1 < 0 || j + 1 >= dim) ? zero : field[IND(i + 1, j + 1, dim)];

    field[IND(i, j, dim)] = (f00 + f01 + f10 + f11 + alpha * b) / beta;
}


__device__ void force(Vector2f x, Vector2f* field, Vector2f c, Vector2f F, float timestep, float r, unsigned dim) {
    float exp = (pow(x(0) - c(0), 2) + pow(x(1) - c(1), 2)) / 2;
    int i = x(0);
    int j = x(1);
    field[IND(i, j, dim)] = F * pow(timestep, exp);
}

__global__ void kernel(Vector2f* u, float* p, float rdx, float viscosity, Vector2f c, Vector2f F, int timestep, float r, unsigned dim)
{
    Vector2f x(threadIdx.x, threadIdx.y);

    //advection
    advect(x, u, u, timestep, rdx, dim);
    __syncthreads(); // barrier
    //diffusion
    float alpha = (rdx * rdx) / (viscosity * timestep);
    float beta = 4 + alpha;
    int i = x(0);
    int j = x(1);
    jacobi<Vector2f>(x, u, alpha, beta, u[IND(i, j, dim)], Vector2f::Zero(), dim);
    __syncthreads();

    //force application
    // apply force every 10 seconds
    if (timestep % 10 == 0)
        force(x, u, c, F, timestep, r, dim);
    __syncthreads();

    //pressure
    alpha = -1 * timestep * timestep;
    beta = 4;
    jacobi<float>(x, p, alpha, beta, divergence(x, u, (float)(rdx / 2), dim), 0, dim);
    __syncthreads();

    // u = w - nabla p
    u[IND(x(0), x(1), dim)] -= gradient(x, p, (float)(rdx / 2), dim);
    __syncthreads(); //potential redundant; implicit barrier between kernel calls
}

int main(void) {
    // quarter of second timestep
    float timestep = 0.25;
    // dimension of vector fields
    unsigned dim = 1024;
    // resolution of display
    unsigned res = 1024;
    // how many pixels a cell of the vector field represents
    float rdx = res / dim;

    // fluid parameters
    float viscosity = 1;

    // force parameters
    Vector2f c((int)(dim / 2), (int)(dim / 2));
    Vector2f F(1, 1);
    float r = 1;

    // half to alloc cpu/ram side u and p, then copy it to device/gpu side u, p.
    // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
    Vector2f *velocity = (Vector2f *)malloc(dim * dim * sizeof(Vector2f));
    float *pressure = (float *)malloc(dim * dim * sizeof(float));

    Vector2f *dev_velocity; // u
    hipMalloc(&dev_velocity, dim * dim * sizeof(Vector2f));
    float *dev_pressure;
    hipMalloc(&dev_pressure, dim * dim * sizeof(float));

    for (int i = 0; i < dim * dim; i++) {
        velocity[i] = Vector2f::Zero();
        pressure[i] = 0;
    }

    hipMemcpy(dev_velocity, velocity, dim*dim*sizeof(Vector2f), hipMemcpyHostToDevice);
    hipMemcpy(dev_pressure, pressure, dim * dim * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(dim, dim);
    while (true) {
        kernel << <1, threads >> > (dev_velocity, dev_pressure, rdx, viscosity, c, F, timestep, r, dim);
        sleep(timestep);
    }
    return 0;
}
